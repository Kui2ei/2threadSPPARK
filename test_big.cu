#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
using namespace std;
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include <cmath>
#include <inttypes.h>
#include <gmp.h>
#include "./bigint.cuh"
#define SIZE 8
float tt=0;

__device__ uint32_t mod[SIZE] = { 
    0xf0000001, 0x43e1f593, 
    0x79b97091, 0x2833e848, 
    0x8181585d, 0xb85045b6, 
    0xe131a029, 0x30644e72};

__global__ void init(){
	M0=computeM0(mod[0]);
}

template<uint32_t limbs>
__global__ void mul_kernel(uint32_t* d_a, uint32_t* d_b, uint32_t* d_c){
	register uint32_t aa[8];
	register uint32_t bb[8];
	register uint32_t cc[8];
	copyArrayValues(aa,d_a);
	copyArrayValues(bb,d_b);
	#pragma unroll
	for(int i=0;i<4096;i++)
    	operator_mul<limbs>(aa,bb,mod,cc,SIZE*32);
	copyArrayValues(d_c,cc);
}

template<uint32_t limbs>
__global__ void add_kernel(uint32_t* d_a, uint32_t* d_b, uint32_t* d_c){
	register uint32_t aa[limbs+1];
	register uint32_t bb[limbs+1];
	register uint32_t cc[8];
	copyArrayValues(aa,d_a);
	copyArrayValues(bb,d_b);
	#pragma unroll
	for(int i=0;i<4096;i++)
		operator_add<limbs>(aa,bb,mod,cc);
	copyArrayValues(d_c,cc);
}

template<uint32_t limbs>
__global__ void sub_kernel(uint32_t* d_a, uint32_t* d_b, uint32_t* d_c){
	register uint32_t aa[limbs+1];
	register uint32_t bb[limbs+1];
	register uint32_t cc[8];
	copyArrayValues(aa,d_a);
	copyArrayValues(bb,d_b);
	#pragma unroll
	for(int i=0;i<4096;i++)
		operator_sub<limbs>(aa,bb,mod,cc);
	copyArrayValues(d_c,cc);
}

void to_mont(uint32_t* data, mpz_t mp_bigint, mpz_t MOD){
	uint32_t temp[SIZE+1]={};
	temp[SIZE]=0x00000001;

	mpz_t mpz_temp;
	mpz_init(mpz_temp);
	mpz_import(mpz_temp, SIZE+1, -1, sizeof(uint32_t), 0, 0, temp);

	mpz_mul(mp_bigint,mp_bigint,mpz_temp);
	mpz_mod(mp_bigint,mp_bigint,MOD);
	size_t mark;
	mpz_export(data, &mark, -1, sizeof(uint32_t), 0, 0, mp_bigint);
	while (mark<SIZE)
		data[mark++] = 0;

}

void mont_back(uint32_t* data, mpz_t mp_bigint, mpz_t MOD){
	uint32_t temp[SIZE+1]={};
	temp[SIZE]=0x00000001;

	mpz_t mpz_temp;
	mpz_init(mpz_temp);
	mpz_import(mpz_temp, SIZE+1, -1, sizeof(uint32_t), 0, 0, temp);
	
	mpz_invert(mpz_temp,mpz_temp,MOD);
	mpz_mul(mp_bigint,mp_bigint,mpz_temp);
	mpz_mod(mp_bigint,mp_bigint,MOD);
	size_t mark;
	mpz_export(data, &mark, -1, sizeof(uint32_t), 0, 0, mp_bigint);
	while (mark<SIZE)
		data[mark++] = 0;

}

int main() {
	// 在主机内存中定义输入数据
	uint32_t a[SIZE];
	uint32_t b[SIZE];
	init<<<1,1>>>();
	
	for(size_t count=0;count<1;count++){
		srand(count);
		for(size_t j=0;j<SIZE;j++){
			a[j]=(rand()<<16)|(rand());
			b[j]=(rand()<<16)|(rand());
		}

		uint32_t c[SIZE] = { 
		0xf0000001, 0x43e1f593, 
		0x79b97091, 0x2833e848, 
		0x8181585d, 0xb85045b6, 
		0xe131a029, 0x30644e72};
		uint32_t r[SIZE];
		uint32_t r_[SIZE];
	
		mpz_t A, B, C, R;

		mpz_init(A); mpz_init(B); mpz_init(C); mpz_init(R);
		mpz_import(A, SIZE, -1, sizeof(uint32_t), 0, 0, a);
		mpz_import(B, SIZE, -1, sizeof(uint32_t), 0, 0, b);
		mpz_import(C, SIZE, -1, sizeof(uint32_t), 0, 0, c);
		for(int i=0;i<64;i++){
			mpz_mul(R, A, B);
			mpz_mod(R, R, C);
		}

		//gmp_printf("Result: %Zd\n", R);
		size_t mark;
		mpz_export(r, &mark, -1, sizeof(uint32_t), 0, 0, R);
		while (mark<SIZE)
			r[mark++] = 0;
		/*
		cout << "GMP:\n";
		//cout<<"MARK:"<<mark<<endl;
		for (int i = 0; i<SIZE; i++)
		{
			cout << hex << r[i] << endl;
		}*/
	
		to_mont(a,A,C);
		to_mont(b,B,C);
	
		uint32_t* d_a;
		uint32_t* d_b;
		uint32_t* d_c;
		hipMalloc(&d_a, sizeof(uint32_t)*SIZE);
		hipMalloc(&d_b, sizeof(uint32_t)*SIZE);
		hipMalloc(&d_c, sizeof(uint32_t)*SIZE);
		hipMemcpy(d_a, a, sizeof(uint32_t)*SIZE, hipMemcpyHostToDevice);
		hipMemcpy(d_b, b, sizeof(uint32_t)*SIZE, hipMemcpyHostToDevice);

		mul_kernel<SIZE> <<<4096,512>>>(d_a, d_b, d_c);
		hipDeviceSynchronize();
		float elapsedTime;
		hipEvent_t e_start, e_stop;
		hipEventCreate(&e_start);
		hipEventCreate(&e_stop);
		hipEventRecord(e_start, 0);
		mul_kernel<SIZE> <<<4096,512>>>(d_a, d_b, d_c);
		hipDeviceSynchronize();
		hipEventRecord(e_stop, 0);
		hipEventSynchronize(e_stop);
		hipEventElapsedTime(&elapsedTime, e_start, e_stop);
		printf("mul time:%.4f ms\n", elapsedTime);
		elapsedTime=0;

		add_kernel<SIZE> <<<4096,512>>>(d_a, d_b, d_c);
		hipDeviceSynchronize();
		hipEventCreate(&e_start);
		hipEventCreate(&e_stop);
		hipEventRecord(e_start, 0);
		add_kernel<SIZE> <<<4096,512>>>(d_a, d_b, d_c);
		hipDeviceSynchronize();
		hipEventRecord(e_stop, 0);
		hipEventSynchronize(e_stop);
		hipEventElapsedTime(&elapsedTime, e_start, e_stop);
		printf("add time:%.4f ms\n", elapsedTime);
		elapsedTime=0;

		sub_kernel<SIZE> <<<4096,512>>>(d_a, d_b, d_c);
		hipDeviceSynchronize();
		hipEventCreate(&e_start);
		hipEventCreate(&e_stop);
		hipEventRecord(e_start, 0);
		sub_kernel<SIZE> <<<4096,512>>>(d_a, d_b, d_c);
		hipDeviceSynchronize();
		hipEventRecord(e_stop, 0);
		hipEventSynchronize(e_stop);
		hipEventElapsedTime(&elapsedTime, e_start, e_stop);
		printf("sub time:%.4f ms\n", elapsedTime);

		//hipMemcpy(r_, d_c, sizeof(uint32_t)*SIZE, hipMemcpyDeviceToHost);

		//mpz_import(A, SIZE, -1, sizeof(uint32_t), 0, 0, r_);
		//mont_back(r_,A,C);
		/*
		cout << "test:\n";
		for (int i = 0; i<SIZE; i++)
		{
			cout << hex << r_[i] << endl;
		}*/
		/*
		for (int i = 0; i<SIZE; i++)
		{

			if(r_[i]!=r[i]){
				printf("error!\n");
				return -1;
			}
		}
		*/
	}
	
	printf("success!\n");
	return 0;
}